#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <cmath>
#include "preconditioner_cuda.h"
#include "spmv_kernel.h"
#include ""
#include "utils.h"  // для convertDenseToCSR

#define CUDA_CHECK(call)                                                      \
    do {                                                                      \
        hipError_t err = call;                                               \
        if(err != hipSuccess) {                                              \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__\
                      << ": " << hipGetErrorString(err) << std::endl;        \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    } while(0)

const double eps = 1e-12;

// --- Таймер на GPU ---
void gpuTimerStart(hipEvent_t* start) {
    hipEventCreate(start);
    hipEventRecord(*start, 0);
}

float gpuTimerStop(hipEvent_t start, hipEvent_t stop) {
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return elapsed;
}

// --- Определения необходимых кернелов ---
// Обновление вектора: p = r + beta*(p - omega*v)
__global__ void updateP(double* p, const double* r, const double* v, double beta, double omega, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        p[i] = r[i] + beta * (p[i] - omega * v[i]);
}

// Вычисление s: s = r - alpha*v
__global__ void computeS(double* s, const double* r, const double* v, double alpha, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        s[i] = r[i] - alpha * v[i];
}

// Обновление x: x = x + alpha*p
__global__ void updateX(double* x, const double* p, double alpha, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        x[i] += alpha * p[i];
}

// Обновление x полностью: x = x + alpha*p + omega*s
__global__ void updateXFull(double* x, const double* p, const double* s, double alpha, double omega, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        x[i] += alpha * p[i] + omega * s[i];
}

// Обновление r: r = s - omega*t
__global__ void updateR(double* r, const double* s, const double* t, double omega, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        r[i] = s[i] - omega * t[i];
}

extern "C" void BiCGStab2_GPU(const double* A, double* x, const double* b,
    int N, double tol, int maxIter, int* iterCount)
{
    // Переводим плотную матрицу в CSR
    double* h_values = nullptr;
    int* h_rowPtr = nullptr;
    int* h_colIdx = nullptr;
    int nnz = 0;
    convertDenseToCSR(A, N, &h_values, &nnz, &h_rowPtr, &h_colIdx);

    // Аллокация CSR на GPU
    double* d_values; int* d_rowPtr; int* d_colIdx;
    CUDA_CHECK(hipMalloc((void**)&d_values, nnz * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_rowPtr, (N + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_colIdx, nnz * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_values, h_values, nnz * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_rowPtr, h_rowPtr, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_colIdx, h_colIdx, nnz * sizeof(int), hipMemcpyHostToDevice));
    delete[] h_values; delete[] h_rowPtr; delete[] h_colIdx;

    // Аллокация векторов на GPU
    double* d_x, * d_b, * d_r, * d_rhat, * d_p, * d_v, * d_s, * d_t, * d_z;
    CUDA_CHECK(hipMalloc((void**)&d_x, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_b, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_r, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_rhat, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_p, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_v, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_s, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_t, N * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_z, N * sizeof(double)));

    CUDA_CHECK(hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice));

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Замер времени на GPU
    hipEvent_t startEvent;
    gpuTimerStart(&startEvent);

    // r = b - A*x. Для SpMV используем наш CSR кернел.
    double* d_temp;
    CUDA_CHECK(hipMalloc(&d_temp, N * sizeof(double)));
    SpMVKernelCSR << <gridSize, blockSize >> > (N, d_values, d_rowPtr, d_colIdx, d_x, d_temp);
    CUDA_CHECK(hipDeviceSynchronize());
    // Вычисляем r = b - y (в данном варианте выполняем передачу на хост, но для производительности нужно убрать лишние hipMemcpy)
    {
        double* h_temp = new double[N];
        double* h_b = new double[N];
        CUDA_CHECK(hipMemcpy(h_temp, d_temp, N * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(h_b, d_b, N * sizeof(double), hipMemcpyDeviceToHost));
        double* h_r = new double[N];
        for (int i = 0; i < N; i++)
            h_r[i] = h_b[i] - h_temp[i];
        CUDA_CHECK(hipMemcpy(d_r, h_r, N * sizeof(double), hipMemcpyHostToDevice));
        delete[] h_temp; delete[] h_b; delete[] h_r;
    }
    CUDA_CHECK(hipFree(d_temp));
    CUDA_CHECK(hipMemcpy(d_rhat, d_r, N * sizeof(double), hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipMemcpy(d_p, d_r, N * sizeof(double), hipMemcpyDeviceToDevice));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double alpha = 1.0, omega = 1.0, rho_old = 1.0;
    int iter = 0;
    double normb = 0.0;
    hipblasDnrm2(handle, N, d_b, 1, &normb);
    if (normb < eps) normb = 1.0;

    while (iter < maxIter) {
        double rho = 0.0;
        hipblasDdot(handle, N, d_rhat, 1, d_r, 1, &rho);
        if (fabs(rho) < tol) break;
        double beta = (iter == 0) ? 0.0 : (rho / (rho_old + eps)) * (alpha / (omega + eps));

        updateP << <gridSize, blockSize >> > (d_p, d_r, d_v, beta, omega, N);
        CUDA_CHECK(hipDeviceSynchronize());

        SpMVKernelCSR << <gridSize, blockSize >> > (N, d_values, d_rowPtr, d_colIdx, d_p, d_v);
        CUDA_CHECK(hipDeviceSynchronize());

        double rhat_dot_v = 0.0;
        hipblasDdot(handle, N, d_rhat, 1, d_v, 1, &rhat_dot_v);
        if (fabs(rhat_dot_v) < tol) break;
        alpha = rho / (rhat_dot_v + eps);

        computeS << <gridSize, blockSize >> > (d_s, d_r, d_v, alpha, N);
        CUDA_CHECK(hipDeviceSynchronize());

        double norm_s = 0.0;
        hipblasDnrm2(handle, N, d_s, 1, &norm_s);
        if (norm_s / normb < tol) {
            updateX << <gridSize, blockSize >> > (d_x, d_p, alpha, N);
            CUDA_CHECK(hipDeviceSynchronize());
            iter++;
            break;
        }

        SpMVKernelCSR << <gridSize, blockSize >> > (N, d_values, d_rowPtr, d_colIdx, d_s, d_t);
        CUDA_CHECK(hipDeviceSynchronize());

        double t_dot_s = 0.0, t_dot_t = 0.0;
        hipblasDdot(handle, N, d_t, 1, d_s, 1, &t_dot_s);
        double temp;
        hipblasDnrm2(handle, N, d_t, 1, &temp);
        t_dot_t = temp * temp;
        if (fabs(t_dot_t) < tol) break;
        omega = t_dot_s / (t_dot_t + eps);

        updateXFull << <gridSize, blockSize >> > (d_x, d_p, d_s, alpha, omega, N);
        CUDA_CHECK(hipDeviceSynchronize());
        updateR << <gridSize, blockSize >> > (d_r, d_s, d_t, omega, N);
        CUDA_CHECK(hipDeviceSynchronize());

        double norm_r = 0.0;
        hipblasDnrm2(handle, N, d_r, 1, &norm_r);
        if (norm_r / normb < tol)
            break;

        rho_old = rho;
        iter++;
    }
    float elapsed_ms = gpuTimerStop(startEvent, 0);
    std::cout << "GPU: Total time = " << elapsed_ms / 1000.0 << " s" << std::endl;
    if (iterCount)
        *iterCount = iter;

    CUDA_CHECK(hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost));

    hipFree(d_values); hipFree(d_rowPtr); hipFree(d_colIdx);
    hipFree(d_x); hipFree(d_b); hipFree(d_r); hipFree(d_rhat);
    hipFree(d_p); hipFree(d_v); hipFree(d_s); hipFree(d_t); hipFree(d_z);

    hipblasDestroy(handle);
}
